#include "paracrine.cuh"
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <cstdlib>


int main() {
	//to do: 
	//convert 
	//create thrust host_vector
	const int nnz = 1024;
	const int grid_size = 32; //grid(grid_size, grid_size, grid_size), assuming cube uniform grid
	thrust::host_vector<Float> neuron_locations_x(nnz+1);
	thrust::host_vector<Float> neuron_locations_y(nnz+1);
	thrust::host_vector<Float> neuron_locations_z(nnz+1);
	thrust::host_vector<Float> neuron_IC(nnz+1);
	thrust::host_vector<Float> grid_IC(grid_size*grid_size*grid_size+1);

	//initialize thrust vector
	//Set neuron locations very simple first (assume 0.5 in x,y,z directions)
	//will eventually need 3 1D vectors
	for (int i = 0; i < nnz; i++){
	neuron_locations_x[i] = 0.5;
	neuron_locations_y[i] = 0.5;
	neuron_locations_z[i] = 0.5;
	}

	//Set initial conditions on grid
	for (int i = 0; i < grid_size * grid_size * grid_size; i++)
		grid_IC[i] = 1;

	//for (int x = 0; x < grid_size; x++) {
	//	for (int y = 0; y < grid_size; y++) {
	//		for (int z = 0; z < grid_size; z++) {
	//			grid_IC[grid_size * grid_size * x + grid_size * y + z]=0;	//grid[grid_height * grid_depth * x + grid_depth * y + z] = grid[x,y,z]
//
//			}
//		}
//	}

	//set some test gridpoints
	//grid_IC[grid_size * grid_size * 0 + grid_size *0 + 0]=1;	//grid[0,0,0]=0
	//grid_IC[grid_size * grid_size * 1 + grid_size *0 + 0]=1;	//grid[1,0,0]=0
	//grid_IC[grid_size * grid_size * 0 + grid_size *1 + 0]=1;	//grid[0,1,0]=0
	//grid_IC[grid_size * grid_size * 1 + grid_size *1 + 0]=1;	//grid[1,1,0]=0
	//grid_IC[grid_size * grid_size * 0 + grid_size *0 + 1]=1;	//grid[0,0,1]=1
	//grid_IC[grid_size * grid_size * 1 + grid_size *0 + 1]=1;	//grid[1,0,1]=1
	//grid_IC[grid_size * grid_size * 0 + grid_size *1 + 1]=1;	//grid[0,1,1]=1
	//grid_IC[grid_size * grid_size * 1 + grid_size *1 + 1]=1;	//grid[1,1,1]=1
	//grid[grid_height * grid_depth * x + grid_depth * y + z] = grid[x,y,z]


	//copy values to host
//	thrust::host_vector<Float> d_neuron_locations = neuron_locations;
//	thrust::host_vector<Float> d_grid_values = grid_values;
	
	//create paracrine object
	paracrine ptest(grid_size, nnz, neuron_locations_x, neuron_locations_y, neuron_locations_z, grid_IC, neuron_IC);

	//initialize paracrine 
	ptest.initialize(); //get Q and weighted_spread vectors to prepare for interpolation, spreading, and diffusion


	//We now have the initial conditions on the grid, so we have to interpolate to the neuron to determine how much neurotransmitter concentration
	//is at the neuron location
//	thrust::device_vector<Float> paracrine_at_neuron(nnz+1);
	thrust::device_vector<Float> d_neuron_conc(nnz);
	d_neuron_conc=ptest.interpolate(nnz,grid_size,grid_IC);
	//grid=ptest.spread(grid_IC, paracrine_at_neuron);

	//copy values from device to host to display them
//	hipMemcpy(thrust::raw_pointer_cast(paracrine_neuron_conc.data()), paracrine_at_neuron_ptr, nnz * sizeof(Float), hipMemcpyDeviceToHost);
//	hipDeviceSynchronize();



	std::cout << "neuron_concentration = " << d_neuron_conc[0] << std::endl;
	//std::cout << "Testing interp and spread" << std::endl;
	//std::cout << grid_IC[0] << std::endl;
	//std::cout << grid[0] << std::endl;



	std::cout << "Spreading Test:" << std::endl;;
	thrust::device_vector<Float> grid(grid_size * grid_size * grid_size+1);
	grid = ptest.spread(grid_IC, d_neuron_conc);
	std::cout << grid[0] << std::endl;






	//Convolution test
	//Set dx
	float dx = 1;
	//Create 3x3x3 discrete laplace stencil (Do this in initialization step, include in paracrine class)
	thrust::device_vector<Float> stencil(27);
	int mask_height = 3;
	int mask_depth = 3;

	//The stencil is flattened using the scheme stencil[x,y,z] = stencil[stencil_height * stencil_depth * x + stencil_depth * y + z];
	//x=0 plane
	stencil[mask_height * mask_depth * 0 + mask_depth * 0 + 0] = 1 / (30 * dx * dx); //stencil[0,0,0]
	stencil[mask_height * mask_depth * 0 + mask_depth * 1 + 0] = 3 / (30 * dx * dx); //stencil[0,1,0]
	stencil[mask_height * mask_depth * 0 + mask_depth * 2 + 0] = 1 / (30 * dx * dx); //stencil[0,2,0]

	stencil[mask_height * mask_depth * 0 + mask_depth * 0 + 1] = 3 / (30 * dx * dx); //stencil[0,0,1]
	stencil[mask_height * mask_depth * 0 + mask_depth * 1 + 1] = 14 / (30 * dx * dx); //stencil[0,1,1]
	stencil[mask_height * mask_depth * 0 + mask_depth * 2 + 1] = 3 / (30 * dx * dx); //stencil[0,2,1]

	stencil[mask_height * mask_depth * 0 + mask_depth * 0 + 2] = 1 / (30 * dx * dx); //stencil[0,0,2]
	stencil[mask_height * mask_depth * 0 + mask_depth * 1 + 2] = 3 / (30 * dx * dx); //stencil[0,1,2]
	stencil[mask_height * mask_depth * 0 + mask_depth * 2 + 2] = 1 / (30 * dx * dx); //stencil[0,2,2]

	//x=1 plane
	stencil[mask_height * mask_depth * 1 + mask_depth * 0 + 0] = 3 / (30 * dx * dx); //stencil[1,0,0]
	stencil[mask_height * mask_depth * 1 + mask_depth * 1 + 0] = 14 / (30 * dx * dx); //stencil[1,1,0]
	stencil[mask_height * mask_depth * 1 + mask_depth * 2 + 0] = 3 / (30 * dx * dx); //stencil[1,2,0]

	stencil[mask_height * mask_depth * 1 + mask_depth * 0 + 1] = 14 / (30 * dx * dx); //stencil[1,0,1]
	stencil[mask_height * mask_depth * 1 + mask_depth * 1 + 1] = -128 / (30 * dx * dx); //stencil[1,1,1]
	stencil[mask_height * mask_depth * 1 + mask_depth * 2 + 1] = 14 / (30 * dx * dx); //stencil[1,2,1]

	stencil[mask_height * mask_depth * 1 + mask_depth * 0 + 2] = 3 / (30 * dx * dx); //stencil[1,0,2]
	stencil[mask_height * mask_depth * 1 + mask_depth * 1 + 2] = 14 / (30 * dx * dx); //stencil[1,1,2]
	stencil[mask_height * mask_depth * 1 + mask_depth * 2 + 2] = 3 / (30 * dx * dx); //stencil[1,2,2]

	//x=2 plane
	stencil[mask_height * mask_depth * 2 + mask_depth * 0 + 0] = 1 / (30 * dx * dx); //stencil[2,0,0]
	stencil[mask_height * mask_depth * 2 + mask_depth * 1 + 0] = 3 / (30 * dx * dx); //stencil[2,1,0]
	stencil[mask_height * mask_depth * 2 + mask_depth * 2 + 0] = 1 / (30 * dx * dx); //stencil[2,2,0]

	stencil[mask_height * mask_depth * 2 + mask_depth * 0 + 1] = 3 / (30 * dx * dx); //stencil[2,0,1]
	stencil[mask_height * mask_depth * 2 + mask_depth * 1 + 1] = 14 / (30 * dx * dx); //stencil[2,1,1]
	stencil[mask_height * mask_depth * 2 + mask_depth * 2 + 1] = 3 / (30 * dx * dx); //stencil[2,2,1]

	stencil[mask_height * mask_depth * 2 + mask_depth * 0 + 2] = 1 / (30 * dx * dx); //stencil[2,0,2]
	stencil[mask_height * mask_depth * 2 + mask_depth * 1 + 2] = 3 / (30 * dx * dx); //stencil[2,1,2]
	stencil[mask_height * mask_depth * 2 + mask_depth * 2 + 2] = 1 / (30 * dx * dx); //stencil[2,2,2]



	//test convolve
	std::cout << "Convolve tester: "<<grid[0] << std::endl;
	grid = ptest.convolve(grid, stencil);
	std::cout << "New  grid value: " << grid[grid_size * grid_size * 20 + grid_size * 20 + 20] << std::endl;

	//Test the padding
	//std::cout << "Image has values: " << std::endl;
	//for (int i=1; i<grid_size+1; i++){
	//	for (int j = 1; j < grid_size + 1; j++) {
	//			std::cout<<image[(grid_size + 2) * i * (grid_size+2) + (grid_size + 2) * (grid_size+1) + j] << std::endl;
	//	}
	//}




}
